#include "hip/hip_runtime.h"
#include<iostream>
#include<cmath>
#include "arrayfire.h"

using namespace std;
#define BLOCK_SIZE 32

//Llenar la matriz A de acuerdo a la formula
//LA MATRIZ A SOLO SE CONTRUYE UNA VEZ POR ESO NO ES 
//NECESARIO PARALELIZARLA
//Llenar la matriz A de acuerdo a la formula
//Como los valores de los bordes ya se conocen, 
//la matriz A solo se construye con los nodos interiores
__global__ void llenarMatrizA_CU(double *A, int nodes, int Nx, int Nz,double sx, double sz){
    double alfa = 1 + (2*sz) + (2*sx);
    int nodo, superior, inferior, derecha, izquierda;
    //Empieza en puntos z = x = 1, porque la cero es una condicion
    //de borde, si se inicia en cero la formula daria indices -1.
    //Lo anterior se aplica tambien para z = Nz -1 y  x = Nx -1
    int j = threadIdx.x + blockIdx.x*blockDim.x;
    if(j > 0 && j < nodes){ // iteterar sobre filas
        //EMPEZAR ITERADORES EN 1, 1 PARA PODER SSABER CUANDO SE VA A 
        //ESTAR EN LOS NODOS DE FRONTERA
        for(int i = 1; i <= Nx; i++){    // iterar sobre columnas
            nodo = ( (j-1)* Nx ) + (i-1);
            superior = nodo + Nx;
            inferior = nodo - Nx;
            derecha = nodo + 1;
            izquierda = nodo - 1;

            //Si i == 1 || i == Nx || j == 1 || j == Nz
            //Son nodos que limitan con los bordes y por lo tanto 
            //van a tener valores de cero

            //Ti+1,j, si i==Nx limita con un borde y por lo tanto el valor no se incluye en A
            if(i != Nx){
                A[((j-1) * nodes ) + derecha] = -sz;
            }
            //Ti,j+1, si j==Nz limita con un borde y por lo tanto el valor no se incluye en A
            if(j != Nz){
                A[((j-1) * nodes ) + superior] = -sx;
            }
            //Ti,j
            A[((j-1) * nodes ) + nodo] = alfa;
            //Ti-1,j, si i==1 limita con un borde y por lo tanto el valor no se incluye en A
            if(i != 1){
                A[((j-1) * nodes ) + izquierda] = -sz;
            }
            //Ti,j-1, si j==1 limita con un borde y por lo tanto el valor no se incluye en A
            if(j != 1){
                A[((j-1) * nodes ) + inferior] = -sx;
            }
        }
    }
}

//Funcion para imprimir solucion
//y graficar con python
void imprimirSolucion(mat &X, int Nx, int Nz, double deltaX, double deltaZ){
    for(double y=0; y< Nz; ++y){
        for(double x=0; x<Nx; ++x){
            //cout<<X(y*Nx+x)<<" ";
            cout<<x*deltaZ<<" "<<y*deltaX<<" "<<X(y*Nx+x)<<endl;
        }
        //cout<<endl;
    }         
}

int main(){

    /* 
        Hacer pruebas con variaciones de deltaX, deltaY
        El calculo y las matrices son de los tamaños interiores
        a los bordes.
    */
    //Variables
    /*  
        temp0: temperatura  a propagar
        x, z: puntos donde empieza la propagacion
    */
    double deltaX,deltaZ,deltaT,k, temp0;
    int Nx,Nz,T,nodos, x, z;
    cin>>deltaX>>deltaZ>>deltaT>>Nx>>Nz>>T>>k>>x>>z;

    //Cantidad de puntos de la malla menos las filas y columnas
    //que componen las condiciones de borde y cuya temperatura es 0
    nodos = Nx * Nz;

    // ARMADILLO CREA LAS MATRICES Y VECTORES POR DEFECTO CON CEROS
    //Matriz tridiagonal A
    //mat A = mat(nodos, nodos);
    double *A = (double*)malloc(nodos * nodos * sizeof(double));
    //Vector B, puntos de la malla con temperaturas conocidas
    //vec B = vec(nodos); 
    double *B = (double*)malloc(nodos * sizeof(double));
    //AX=B;
    //vec X = vec(nodos);
    double *X = (double*)malloc(nodos * sizeof(double));
    // Condición inicial, lugar(x,z) de la malla donde se pone la temperatura tem0
    // x -> derecha - izquierda : horizontal
    // z -> arriba - abajo : vertial
    X[(Nx * z + x)] = temp0;
    double sx = (k*deltaT)/(deltaX*deltaX);
    double sy = (k*deltaT)/(deltaZ*deltaZ);

    double *d_A;
    error = hipMalloc((void**)&d_A, nodos* nodos * sizeof(double));
    if (error != hipSuccess) {
        printf("Error allocating memory to d_A");
        return 1;
    }

    hipMemcpy(d_A, A, nodes * nodes * sizeof(float), hipMemcpyHostToDevice);
    int blockSize = 32;
	dim3 dimblock(blockSize, blockSize, 1);
    dim3 dimGrid(ceil((nodos) / float(blockSize)), ceil((nodos) / float(blockSize)), 1);
    time_t start, end;
    double timeGPU;
    start = clock();
	llenarMatrizA_CU<<<dimGrid,dimblock>>>(d_A, nodos, Nx, Nz, sx, sz);
	hipDeviceSynchronize();
    end = clock();

    timeGPU = difftime(end, start);
    printf ("Elasped time in GPU: %.2lf seconds.\n", timeGPU);
    
    hipMemcpy(A, d_A, nodos * nodos * sizeof(double), hipMemcpyDeviceToHost);
    //llenarMatrizA(A, Nx, Nz, sx, sy);


    /* CODIGO ARRAYFIRE */
    //Se debe especificar la GPU del computador
    int device = 0;
    af::setDevice(device);
    // af::info();

    //Variables para pasar matriz de armadillo a arrayfire
    // double *A_mem = (double*)malloc(nodos*nodos*sizeof(double));
    // double *B_mem = (double*)malloc(nodos*sizeof(double));
    // double *X_mem = (double*)malloc(nodos*sizeof(double));
    //Pasar matrices a punteros de c++
    // A_mem = A.memptr();
    // B_mem = B.memptr();
    // X_mem = X.memptr();
    //Creacion de arrays en ArrayFire
    af::array afA(nodos,nodos,A);
    af::array afB(nodos,B);
    af::array afX(nodos,B;

    //Poner temperatura inicial

    //Calculo de temperatura de la malla para cada tiempo
    for(int t = 0; t < T-1; t++){
        //En el tiempo 0, X contiene la malla con las
        //condiciones iniciales
        afB = afX;
        //Solucion del sistema de ecuaciones usando ArrayFire
        //X contiene la temperatura en el tiempo t   
        afX = af::solve(afA,afB);
    }
    return 0;
}

